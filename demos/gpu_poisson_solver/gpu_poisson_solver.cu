#include "hip/hip_runtime.h"
// Copyright © 2016,2017 Ryabkov Oleg Igorevich, Evstigneev Nikolay Mikhaylovitch

// This file is part of SimpleCFD.

// SimpleCFD is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, version 2 only of the License.

// SimpleCFD is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.

// You should have received a copy of the GNU General Public License
// along with SimpleCFD.  If not, see <http://www.gnu.org/licenses/>.

//TODO include cuda explicitly

#include "gpu_poisson_solver_config.h"
#include <string>
#include <cmath>
#include <scfd/utils/log_std.h>
#include <scfd/utils/main_try_catch_macro.h>
#include <scfd/utils/init_cuda.h>
#include <scfd/utils/constant_data.h>
#include <scfd/static_vec/vec.h>
#include <scfd/memory/hip/hip_runtime.h>
#include <scfd/communication/linear_partitioner.h>
//TODO add serial_map
#include <scfd/communication/serial_map.h>
#include <scfd/mesh/gmsh_mesh_wrap.h>
#include <scfd/mesh/host_mesh.h>
#include <scfd/mesh/device_mesh.h>
#include <scfd/mesh/device_mesh_impl.h>
#include "gmsh_pos_output.h"

using real = GPU_POISSON_SOLVER_SCALAR_TYPE;
using ordinal = int;
static const ordinal dim = 3;
using partitioner_t = scfd::communication::linear_partitioner;
using gmsh_wrap_t = scfd::mesh::gmsh_mesh_wrap<real,partitioner_t,dim,ordinal>;
using host_mesh_t = scfd::mesh::host_mesh<gmsh_wrap_t>;
using vec_t = scfd::static_vec::vec<real,dim>;
using log_t = scfd::utils::log_std;
using mem_t = scfd::memory::cuda_device;
using device_mesh = scfd::mesh::device_mesh<real,mem_t,dim,ordinal>;

typedef t_serial_map            t_map;
typedef t_tensor0_field_tml<real>   t_vars;

DEFINE_CONSTANT_BUFFER(t_gpu_mesh, mesh)

__device__ int  get_elem_type(int i)    //gmsh element type
{
    if (mesh().is_homogeneous) return mesh().homogeneous_elem_type; else return mesh().elem_type(i,0);
}

//these service function related to mesh reference information could be placed in gpu_mesh
__device__ int  get_elem_faces_n(int elem_type)
{
    //ISSUE to make faster access (perhaps, not tested yet) we can make constant array
    if (elem_type == 4) return 4;
    if (elem_type == 5) return 6;
    if (elem_type == 6) return 5;
    if (elem_type == 7) return 5;
    //TODO others, error
    return 0;   //just to remove warning
}

__device__ int  get_elem_vert_n(int elem_type)
{
    //ISSUE to make faster access (perhaps, not tested yet) we can make constant array
        if (elem_type == 4) return 4;
        if (elem_type == 5) return 8;
        if (elem_type == 6) return 6;
        if (elem_type == 7) return 5;
        //TODO others, error
        return 0;   //just to remove warning
}

//returns p0 reflected with respect to plane with normal norm and point p1 on it
__device__ t_vec reflect_point(const t_vec &norm, const t_vec &p1, const t_vec &p0)
{
    real d = -scalar_prod(norm,p1);
    //could be of any sign
    real    dest = std::abs(scalar_prod(norm,p0) + d);
    return p0 + norm*(real(2.f)*dest);
}

__global__ void ker_poisson_iteration(t_vars vars_old, t_vars vars_new, int bnd1, int bnd2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!((i >= 0)&&(i < mesh().n_cv))) return;

        int elem_type = get_elem_type(i);
    real    numerator(0.f), denominator(0.f);
    for (int j = 0;j < get_elem_faces_n(elem_type);++j) {
        int nb = mesh().Neighbour(i,j);
        t_vec   nb_center;
        real    dist, var_nb;
        if (nb != CUDA_EMPTY_IDX) {
            mesh().center_neighbour.getv(i,j,nb_center);
            var_nb = vars_old(nb);
        } else {
            nb_center = reflect_point(mesh().Norm.getv(i,j), mesh().center_faces.getv(i,j), mesh().center.getv(i));
            if (mesh().Boundary(i,j) == bnd1) {
                //dirichle 0. value
                var_nb = -vars_old(i);
            } else if (mesh().Boundary(i,j) == bnd2) {
                //dirichle 1. value
                var_nb = real(2.f)*real(1.f)-vars_old(i);
            } else {
                //neumann
                var_nb = vars_old(i);
            }
        }
        dist = scalar_prod(mesh().Norm.getv(i,j), nb_center - mesh().center.getv(i));

        numerator += mesh().faces_S(i,j)*var_nb/dist;
        denominator += mesh().faces_S(i,j)/dist;
    }
    vars_new(i) = numerator/denominator;
}

//B := A
__global__ void ker_fill_zero(t_vars A)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!((i >= 0)&&(i < mesh().n_cv))) return;
        A(i) = real(0.f);
}

//B := A
__global__ void ker_assign(t_vars B, t_vars A)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!((i >= 0)&&(i < mesh().n_cv))) return;
        B(i) = A(i);
}

int main(int argc, char **args)
{
    LogStd          log;
    t_cpu_mesh      cpu_mesh;
        t_map           map;
    t_gpu_mesh      gpu_mesh;
    t_vars          vars0, vars1;
    int         device_number;
    int         bnd1, bnd2, iters_num;
    dim3            dimBlock, dimGrid;

    USE_MAIN_TRY_CATCH(log)

    log.set_verbosity(1);

    //process args
    if (argc < 5) {
        printf("usage: gpu_poisson_solver device_number bnd1_id bnd2_id iters_num\n");
        printf("    mesh is read from 'mesh.dat' file\n");
        printf("example: ./gpu_poisson_solver 0 6 166 1000\n");
        return 1;
    }
    device_number = atoi(args[1]);
    bnd1 = atoi(args[2]);
    bnd2 = atoi(args[3]);
    iters_num = atoi(args[4]);

    MAIN_TRY("reading mesh")
        if (!cpu_mesh.read("mesh.dat")) throw std::runtime_error("failed to read mesh from mesh.dat");
        //init map object
        map = t_serial_map(cpu_mesh.cv.size());
        //TODO add 0th order stencil through add_stencil_element()
        map.complete();
    MAIN_CATCH(2)

    MAIN_TRY("init CUDA")
        if (!InitCUDA(device_number)) throw std::runtime_error("InitCUDA failed");
    MAIN_CATCH(3)

    MAIN_TRY("allocate memory for mesh in device")
        gpu_mesh.init(map);
    MAIN_CATCH(4)

    MAIN_TRY("copy mesh data to device")
        init_gpu_mesh(map, gpu_mesh, cpu_mesh);
        //copy info about gpu mesh to gpu constant buffer
                COPY_TO_CONSTANT_BUFFER(mesh, gpu_mesh);
    MAIN_CATCH(5)
    
    dimBlock = dim3(128);
    dimGrid = dim3((gpu_mesh.n_cv+128)/dimBlock.x);

    MAIN_TRY("allocating variables array")
        //TODO we could make cool init using MAP concept, like init init_local methods
        vars0.init(map.max_loc_ind() - map.min_loc_ind() + 1, map.min_loc_ind());
        vars1.init(map.max_loc_ind() - map.min_loc_ind() + 1, map.min_loc_ind());
        MAIN_CATCH(6)

        MAIN_TRY("iterate poisson equation")
        ker_fill_zero<<<dimGrid, dimBlock>>>(vars0);
        ker_fill_zero<<<dimGrid, dimBlock>>>(vars1);
            for (int i = 0;i < iters_num;++i) {
            log.info_f("iteration %d", i);
            //put result in vars1
            ker_poisson_iteration<<<dimGrid, dimBlock>>>(vars0, vars1, bnd1, bnd2);
            //vars0 := vars1
                        ker_assign<<<dimGrid, dimBlock>>>(vars0, vars1);
        }
        MAIN_CATCH(7)
        
        MAIN_TRY("writing pos output to result.pos")
            write_out_pos_scalar_file("result.pos", "poisson_phi", cpu_mesh, map, vars0);
        MAIN_CATCH(8)

        //NOTE memory deallocation done in destrcutor automatically

    return 0;
}
