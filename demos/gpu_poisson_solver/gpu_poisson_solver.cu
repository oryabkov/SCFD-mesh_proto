#include "hip/hip_runtime.h"
// Copyright © 2016,2017 Ryabkov Oleg Igorevich, Evstigneev Nikolay Mikhaylovitch

// This file is part of SimpleCFD.

// SimpleCFD is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, version 2 only of the License.

// SimpleCFD is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.

// You should have received a copy of the GNU General Public License
// along with SimpleCFD.  If not, see <http://www.gnu.org/licenses/>.

//TODO include cuda explicitly

//#define SCFD_MAIN_TRY_CATCH_DISABLE_CATCH 1

#include "gpu_poisson_solver_config.h"
#include <string>
#include <cmath>
#include <scfd/utils/log_std.h>
#include <scfd/utils/main_try_catch_macro.h>
#include <scfd/utils/init_cuda.h>
#include <scfd/utils/constant_data.h>
#include <scfd/static_vec/vec.h>
#include <scfd/memory/hip/hip_runtime.h>
#include <scfd/for_each/hip/hip_runtime.h>
#include <scfd/for_each/cuda_impl.cuh>
#include <scfd/communication/linear_partitioner.h>
//TODO add serial_map
#include <scfd/communication/serial_map.h>
#include <scfd/mesh/gmsh_mesh_wrap.h>
#include <scfd/mesh/host_mesh.h>
#include <scfd/mesh/device_mesh.h>
#include <scfd/mesh/device_mesh_impl.h>
#include "gmsh_pos_output.h"
#include "map_mock.h"

using real = GPU_POISSON_SOLVER_SCALAR_TYPE;
using ordinal = int;
static const ordinal dim = 3;
using partitioner_t = scfd::communication::linear_partitioner;
using gmsh_wrap_t = scfd::mesh::gmsh_mesh_wrap<real,partitioner_t,dim,ordinal>;
using host_mesh_t = scfd::mesh::host_mesh<gmsh_wrap_t>;
using vec_t = scfd::static_vec::vec<real,dim>;
using log_t = scfd::utils::log_std;
using mem_t = scfd::memory::cuda_device;
using device_mesh_t = scfd::mesh::device_mesh<real,mem_t,dim,ordinal>;
using host_real_vector_t = std::vector<real>;
using for_each_t = scfd::for_each::cuda<>;

SCFD_DEVICE_MESH_INSTANTIATE(real,mem_t,dim,ordinal)

typedef scfd::communication::serial_map  map_t;
typedef scfd::arrays::array<real,mem_t>  vars_t;

DEFINE_CONSTANT_BUFFER(device_mesh_t, mesh)

__device__ int  get_elem_type(int i)    //gmsh element type
{
    if (mesh().is_homogeneous) return mesh().homogeneous_elem_type; else return mesh().elems_types(i);
}

//these service function related to mesh reference information could be placed in gpu_mesh
__device__ int  get_elem_faces_n(int elem_type)
{
    //ISSUE to make faster access (perhaps, not tested yet) we can make constant array
    if (elem_type == 4) return 4;
    if (elem_type == 5) return 6;
    if (elem_type == 6) return 5;
    if (elem_type == 7) return 5;
    //TODO others, error
    return 0;   //just to remove warning
}

__device__ int  get_elem_vert_n(int elem_type)
{
    //ISSUE to make faster access (perhaps, not tested yet) we can make constant array
        if (elem_type == 4) return 4;
        if (elem_type == 5) return 8;
        if (elem_type == 6) return 6;
        if (elem_type == 7) return 5;
        //TODO others, error
        return 0;   //just to remove warning
}

//returns p0 reflected with respect to plane with normal norm and point p1 on it
__device__ vec_t reflect_point(const vec_t &norm, const vec_t &p1, const vec_t &p0)
{
    real d = -scalar_prod(norm,p1);
    //could be of any sign
    real    dest = std::abs(scalar_prod(norm,p0) + d);
    return p0 + norm*(real(2.f)*dest);
}

struct bnd_cond_data_t
{
    ordinal     dirichle_bnds_num;
    ordinal     dirichle_bnds[10];
    real        dirichle_vals[10];
};

DEFINE_CONSTANT_BUFFER(bnd_cond_data_t, bnd_cond_data)

struct force_data_t
{
    /// amplitude
    real    a;
    /// space multipliers
    real    omega[dim];
    /// space phase shifts
    real    phi[dim];
};

DEFINE_CONSTANT_BUFFER(force_data_t, force_data)

__global__ void ker_poisson_iteration(vars_t vars_old, vars_t vars_new, int bnd1, int bnd2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!((i >= mesh().own_elems_range.i0)&&(i < mesh().own_elems_range.i0 + mesh().own_elems_range.n))) return;

    int     elem_type = get_elem_type(i);
    real    numerator(0.f), denominator(0.f);
    for (int j = 0;j < get_elem_faces_n(elem_type);++j) {
        int nb = mesh().elems_neighbours0(i,j);
        vec_t   nb_center;
        real    dist, var_nb;
        if (nb != device_mesh_t::special_id) {
            mesh().elems_neighbours0_centers.get_vec(nb_center,i,j);
            var_nb = vars_old(nb);
        } else {
            nb_center = reflect_point(mesh().elems_faces_norms.get_vec(i,j), mesh().elems_faces_centers.get_vec(i,j), mesh().elems_centers.get_vec(i));
            //real x = mesh().elems_faces_centers(i,j,0);
            ordinal  bnd_group_id = mesh().elems_faces_group_ids(i,j);
            bool     dirichle_found = false;
            real     dirichle_val;
            for (ordinal bnd_i = 0;bnd_i < bnd_cond_data().dirichle_bnds_num;++bnd_i)
            {
                if (bnd_group_id == bnd_cond_data().dirichle_bnds[bnd_i])
                {
                    dirichle_found = true;
                    dirichle_val = bnd_cond_data().dirichle_vals[bnd_i];
                    break;
                }
            }
            if (dirichle_found) {
                //dirichle value
                var_nb = real(2.f)*dirichle_val-vars_old(i);
            } else {
                //neumann
                var_nb = vars_old(i);
            }
            //var_nb = real(2.f)*x-vars_old(i);
        }
        dist = scalar_prod(mesh().elems_faces_norms.get_vec(i,j), nb_center - mesh().elems_centers.get_vec(i));

        numerator += mesh().elems_faces_areas(i,j)*var_nb/dist;
        denominator += mesh().elems_faces_areas(i,j)/dist;
    }
    vars_new(i) = numerator/denominator;
    //real x = mesh().elems_centers(i,0);
    //vars_new(i) = x;
}

//B := A
__global__ void ker_fill_zero(vars_t A)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!((i >= mesh().own_elems_range.i0)&&(i < mesh().own_elems_range.i0 + mesh().own_elems_range.n))) return;
    A(i) = real(0.f);
}

//B := A
__global__ void ker_assign(vars_t B, vars_t A)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (!((i >= mesh().own_elems_range.i0)&&(i < mesh().own_elems_range.i0 + mesh().own_elems_range.n))) return;
    B(i) = A(i);
}

void read_bnd_data(const std::string &fn, bnd_cond_data_t &res, std::set<ordinal> &periodic_bnds)
{
    std::ifstream   f(fn.c_str());
    ordinal         bnds_n;
    if (!(f >> bnds_n))
        throw std::runtime_error("read_bnd_data: failed to read from file " + fn);
    res.dirichle_bnds_num = 0;
    for (ordinal bnd_i = 0;bnd_i < bnds_n;++bnd_i)
    {
        ordinal         bnd_id;
        std::string     bnd_type;
        real            bnd_val;
        if (!(f >> bnd_id >> bnd_type >> bnd_val))
            throw std::runtime_error("read_bnd_data: failed to read from file " + fn);

        if (bnd_type == "D")
        {
            res.dirichle_bnds[res.dirichle_bnds_num] = bnd_id;
            res.dirichle_vals[res.dirichle_bnds_num] = bnd_val;
            ++res.dirichle_bnds_num;
            if (res.dirichle_bnds_num > 10)
                throw std::runtime_error("read_bnd_data: res.dirichle_bnds_num > 10");
        } 
        else if (bnd_type == "P")
        {
            periodic_bnds.insert(bnd_id);
        }
        else
        {
            throw std::runtime_error("read_bnd_data: unknown boundary type " + bnd_type);
        }
    }
}

force_data_t read_force_data(const std::string &fn)
{
    force_data_t res;

    std::ifstream   f(fn.c_str());
    if (!(f >> res.a))
        throw std::runtime_error("read_force_data: failed to read from file " + fn);
    for (int j = 0;j < dim;++j)
    {
        if (!(f >> res.omega[j] >> res.phi[j]))
            throw std::runtime_error("read_force_data: failed to read from file " + fn);
    }

    return res;
}

int main(int argc, char **args)
{
    std::string         mesh_fn, bnd_fn, force_fn;
    int                 device_number;
    ordinal             iters_num;
    std::set<ordinal>   periodic_bnds;

    log_t               log;
    auto                part = std::make_shared<partitioner_t>();
    auto                host_mesh = std::make_shared<host_mesh_t>();

    auto                map = std::make_shared<map_t>();
    device_mesh_t       gpu_mesh;
    host_real_vector_t  vars_host;
    vars_t              vars0, vars1;
    dim3                dimBlock, dimGrid;

    USE_MAIN_TRY_CATCH(log)

    log.set_verbosity(1);

    //process args
    if (argc < 6)
    {
        printf("Usage: ./gpu_poisson_solver.bin DEVICE_NUMBER MESH_FN BNDS_FN FORCE_FN ITERS_NUM\n");
        printf("Example: ./gpu_poisson_solver.bin 0 test.msh 5 27 1000\n");
        return 1;
    }
    device_number = atoi(args[1]);
    mesh_fn = args[2];
    bnd_fn = args[3];
    force_fn = args[4];
    iters_num = atoi(args[5]);

    MAIN_TRY("reading boundary data from from " + bnd_fn)
    bnd_cond_data_t bnd_cond_data_host;
    read_bnd_data(bnd_fn, bnd_cond_data_host, periodic_bnds);
    COPY_TO_CONSTANT_BUFFER(bnd_cond_data, bnd_cond_data_host);
    MAIN_CATCH(2)

    MAIN_TRY("reading force data from from " + force_fn)
    force_data_t force_data_host = read_force_data(force_fn);
    COPY_TO_CONSTANT_BUFFER(force_data, force_data_host);
    MAIN_CATCH(2)

    MAIN_TRY("reading mesh from " + mesh_fn)
    host_mesh->set_mesh_filename(mesh_fn);
    host_mesh->read(periodic_bnds);
    *part = partitioner_t(host_mesh->get_total_elems_num(), 1, 0);
    host_mesh->set_partitioner(part);
    host_mesh->enlarge_stencil(1);
    //init map object
    *map = map_t(host_mesh->get_total_elems_num());
    //TODO add 0th order stencil through add_stencil_element()
    map->complete();
    MAIN_CATCH(2)

    MAIN_TRY("init CUDA")
    //TODO
    //if (!InitCUDA(device_number)) throw std::runtime_error("InitCUDA failed");
    scfd::utils::init_cuda(-2, 0);
    MAIN_CATCH(3)

    MAIN_TRY("allocate memory for mesh in device and copy mesh data to device")
    for_each_t  for_each;
    gpu_mesh.params.has_elems_nodes_data = false;
    gpu_mesh.init_elems_data
    (
        *host_mesh, *map, map_mock(), map_mock(), for_each
    );
    COPY_TO_CONSTANT_BUFFER(mesh, gpu_mesh);
    MAIN_CATCH(4)
    
    //TODO
    dimBlock = dim3(128);
    dimGrid = dim3((gpu_mesh.own_elems_range.i1() + dimBlock.x)/dimBlock.x);

    MAIN_TRY("allocating variables array")
    //TODO we could make cool init using MAP concept, like init init_local methods
    //TODO
    vars0.init(map->max_loc_ind() - map->min_loc_ind() + 1, map->min_loc_ind());
    vars1.init(map->max_loc_ind() - map->min_loc_ind() + 1, map->min_loc_ind());
    vars_host = host_real_vector_t(map->get_total_size());
    MAIN_CATCH(5)

    MAIN_TRY("iterate poisson equation")
    ker_fill_zero<<<dimGrid, dimBlock>>>(vars0);
    ker_fill_zero<<<dimGrid, dimBlock>>>(vars1);
    for (int i = 0;i < iters_num;++i) 
    {
        log.info_f("iteration %d", i);
        //put result in vars1
        ker_poisson_iteration<<<dimGrid, dimBlock>>>(vars0, vars1, bnd1, bnd2);
        //vars0 := vars1
        ker_assign<<<dimGrid, dimBlock>>>(vars0, vars1);
    }
    MAIN_CATCH(6)

    MAIN_TRY("copy results to host")
    auto                    vars0_view = vars0.create_view(true);
    //auto                    vars0_view = gpu_mesh.elems_vols.create_view(true);
    for (ordinal i_ = 0;i_ < map->get_size();++i_) 
    {
        ordinal         i_glob = map->own_glob_ind(i_);
        ordinal         i = map->own_loc_ind(i_);
        vars_host[i_glob] = vars0_view(i);
        //vars_host[i_glob] = vars0_view(i,0);
    }
    vars0_view.release(false);
    MAIN_CATCH(7)
    
    MAIN_TRY("writing pos output to result.pos")
    write_out_pos_scalar_file("result.pos", "poisson_phi", *host_mesh, vars_host);
    MAIN_CATCH(8)

    //NOTE memory deallocation done in destrcutor automatically

    return 0;
}
