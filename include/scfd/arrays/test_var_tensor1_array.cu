
#include <scfd/utils/init_cuda.h>
#include <scfd/memory/host.h>
#include <scfd/memory/hip/hip_runtime.h>
#include <scfd/arrays/var_tensor1_array.h>

using namespace scfd;

int main(int argc, char const *args[])
{
    arrays::var_tensor1_array<double,memory::cuda_host>     host_arr;
    arrays::var_tensor1_array<double,memory::cuda_device>   cuda_arr;


    
    return 0;
}